#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <opencv2/core.hpp>
// #include <opencv2/highgui/highgui.hpp>
#include <chrono> 

#include <opencv4/opencv2/core/core_c.h>
#include <opencv2/highgui/highgui_c.h>
#include <hip/hip_runtime.h>
// #include <crt/hip/device_functions.h>
#include <>
#include <hip/hip_complex.h>
#include <opencv4/opencv2/imgcodecs.hpp>
// #include <opencv2/core/mat.hpp>
// #include <opencv2>s


using namespace std::chrono;
using namespace std;
using namespace cv;

typedef complex<float> base;
typedef float2 Complex_my;

template <typename T>
ostream &operator<<(ostream &o, vector<T> v)
{
    if (v.size() > 0)
        o << v[0];
    for (unsigned i = 1; i < v.size(); i++)
        o << " " << v[i];
    return o << endl;
}
static __device__ __host__ inline Complex_my Add(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x + B.x;
    C.y = A.y + B.y;
    return C;
}

/**
 *  Inverse of Complex_my Number
 */
static __device__ __host__ inline Complex_my Inverse(Complex_my A)
{
    Complex_my C;
    C.x = -A.x;
    C.y = -A.y;
    return C;
}

/**
 *  Multipication of Complex_my Numbers
 */
static __device__ __host__ inline Complex_my Multiply(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x * B.x - A.y * B.y;
    C.y = A.y * B.x + A.x * B.y;
    return C;
}

/**
* Parallel Functions for performing various tasks
*/

/**
*  Dividing by constant for inverse fft transform
*/
__global__ void inplace_divide_invert(Complex_my *A, int n, int threads)
{
    int i = blockIdx.x * threads + threadIdx.x;
    if (i < n)
    {
        // printf("in divide");
        A[i].x /= n;
        A[i].y /= n;
    }
    else
    {
        // printf("else in divide");
        // printf("i=%d, n=%d", i, n);
    }
}

/**
* Reorders array by bit-reversing the indexes.
*/
__global__ void bitrev_reorder(Complex_my *__restrict__ r, Complex_my *__restrict__ d, int s, size_t nthr, int n)
{
    int id = blockIdx.x * nthr + threadIdx.x;
    //r[id].x = -1;
    if (id < n and __brev(id) >> (32 - s) < n)
        r[__brev(id) >> (32 - s)] = d[id];
}

/**
* Inner part of the for loop
*/
__device__ void inplace_fft_inner(Complex_my *__restrict__ A, int i, int j, int len, int n, bool invert)
{
    if (i + j + len / 2 < n and j < len / 2)
    {
        Complex_my u, v;

        float angle = (2 * M_PI * j) / (len * (invert ? 1.0 : -1.0));
        v.x = cos(angle);
        v.y = sin(angle);

        u = A[i + j];
        v = Multiply(A[i + j + len / 2], v);
        // printf("i:%d j:%d u_x:%f u_y:%f    v_x:%f v_y:%f\n", i, j, u.x, u.y, v.x, v.y);
        A[i + j] = Add(u, v);
        A[i + j + len / 2] = Add(u, Inverse(v));
    }
}

// CUDA Kernel function to generate the Hanning window
__global__ void hanningWindow(float *d_in, int N, float scale_factor = 1.0) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float scale = 2.0f * 3.14159265358979323846f / (N - 1);
        d_in[idx] *= scale_factor * 0.5f * (1.0f - cosf(scale * idx));
    }
}

/**
* FFT if number of threads are sufficient.
*/
__global__ void inplace_fft(Complex_my *__restrict__ A, int i, int len, int n, int threads, bool invert)
{
    int j = blockIdx.x * threads + threadIdx.x;
    inplace_fft_inner(A, i, j, len, n, invert);
}

/**
* FFt if number of threads are not sufficient.
*/
__global__ void inplace_fft_outer(Complex_my *__restrict__ A, int len, int n, int threads, bool invert)
{
    int i = (blockIdx.x * threads + threadIdx.x)*len;
    for (int j = 0; j < len / 2; j++)
    {
        inplace_fft_inner(A, i, j, len, n, invert);
    }
}

/**
* parallel FFT transform and inverse transform
* Arguments vector of complex numbers, invert, balance, number of threads
* Perform inplace transform
*/
void fft(vector<base> &a, bool invert, int balance = 10, int threads = 32)
{
    // Creating array from vector
    int n = (int)a.size();
    int data_size = n * sizeof(Complex_my);
    Complex_my *data_array = (Complex_my *)malloc(data_size);
    for (int i = 0; i < n; i++)
    {
        data_array[i].x = a[i].real();
        data_array[i].y = a[i].imag();
    }
    
    // Copying data to GPU
    Complex_my *A, *dn;
    hipMalloc((void **)&A, data_size);
    hipMalloc((void **)&dn, data_size);
    hipMemcpy(dn, data_array, data_size, hipMemcpyHostToDevice);
    // Bit reversal reordering
    int s = log2(n);

    bitrev_reorder<<<ceil(float(n) / threads), threads>>>(A, dn, s, threads, n);

    
    // Synchronize
    hipDeviceSynchronize();
    // Iterative FFT with loop parallelism balancing
    for (int len = 2; len <= n; len <<= 1)
    {
        if (n / len > balance)
        {

            inplace_fft_outer<<<ceil((float)n / threads / len), threads>>>(A, len, n, threads, invert);
        }
        else
        {
            for (int i = 0; i < n; i += len)
            {
                float repeats = len / 2;
                inplace_fft<<<ceil(repeats / threads), threads>>>(A, i, len, n, threads, invert);
            }
        }
    }
    
    if (invert)
        inplace_divide_invert<<<ceil(n * 1.00 / threads), threads>>>(A, n, threads);

    // Copy data from GPU
    Complex_my *result;
    result = (Complex_my *)malloc(data_size);
    hipMemcpy(result, A, data_size, hipMemcpyDeviceToHost);
    
    // Saving data to vector<complex> in input.
    for (int i = 0; i < n; i++)
    {
        a[i] = base(result[i].x, result[i].y);
    }
    // Free the memory blocks
    free(data_array);
    hipFree(A);
    hipFree(dn);
    return;
}

/** Applying hanning window*/
void applyHanningWindow(vector<base> &a) {
    float *d_in;
    hipMalloc(&d_in, a.size() * sizeof(base));
    hipMemcpy(d_in, a.data(), a.size() * sizeof(base), hipMemcpyHostToDevice);

    hanningWindow<<<(a.size() + 255) / 256, 256>>>(d_in, a.size(), 0.7);
    hipDeviceSynchronize();

    hipMemcpy(a.data(), d_in, a.size() * sizeof(base), hipMemcpyDeviceToHost);
    hipFree(d_in);
}

/**
* Performs 2D FFT 
* takes vector of complex vectors, invert and verbose as argument
* performs inplace FFT transform on input vector
*/
void fft2D(vector<vector<base>> &a, bool invert, int balance, int threads, int verbose = 0)
{
    auto matrix = a;
    // Transform the rows
    if (verbose > 0)
        cout << "Transforming Rows" << endl;

    for (auto i = 0; i < matrix.size(); i++)
    {
        //cout<<i<<endl;
        applyHanningWindow(matrix[i]);
        fft(matrix[i], invert, balance, threads);
    }

    // preparing for transforming columns

    if (verbose > 0)
        cout << "Converting Rows to Columns" << endl;

    a = matrix;
    matrix.resize(a[0].size());
    for (int i = 0; i < matrix.size(); i++)
        matrix[i].resize(a.size());

    // Transposing matrix
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            matrix[j][i] = a[i][j];
        }
    }
    if (verbose > 0)
        cout << "Transforming Columns" << endl;

    // Transform the columns
    for (auto i = 0; i < matrix.size(); i++){
        applyHanningWindow(matrix[i]);
        fft(matrix[i], invert, balance, threads);
    }

    if (verbose > 0)
        cout << "Storing the result" << endl;

    // Storing the result after transposing
    // [j][i] is getting value of [i][j]
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            a[j][i] = matrix[i][j];
        }
    }
}

/**
* Function to multiply two polynomial
* takes two polynomials represented as vectors as input
* return the product of two vectors
*/
vector<int> mult(vector<int> a, vector<int> b, int balance, int threads)
{
    // Creating complex vector from input vectors
    vector<base> fa(a.begin(), a.end()), fb(b.begin(), b.end());

    // Padding with zero to make their size equal to power of 2
    size_t n = 1;
    while (n < max(a.size(), b.size()))
        n <<= 1;
    n <<= 1;

    fa.resize(n), fb.resize(n);

    // Transforming both a and b
    // Converting to points form
    fft(fa, false, balance, threads), fft(fb, false, balance, threads);

    // performing point wise multipication of points
    for (size_t i = 0; i < n; ++i)
        fa[i] *= fb[i];

    // Performing Inverse transform
    fft(fa, true, balance, threads);

    // Saving the real part as it will be the result
    vector<int> res;
    res.resize(n);
    for (size_t i = 0; i < n; ++i)
        res[i] = int(fa[i].real() + 0.5);

    return res;
}

/**
* Function to perform jpeg compression on image
* takes image, threshold, verbose as input
* image is represented as vector<vector>
* perform inplace compression on the input
*/
void compress_image(vector<vector<uint>> &image, double threshold, int balance, int threads, int verbose = 1)
{
    //Convert image to complex type
    cout << "here";
    vector<vector<base>> complex_image(image.size(), vector<base>(image[0].size()));
    for (auto i = 0; i < image.size(); i++)
    {
        for (auto j = 0; j < image[0].size(); j++)
        {
            complex_image[i][j] = image[i][j];
        }
    }
    if (verbose == 1)
    {
        cout << "input Image" << endl;
        //cout << image;
        cout << endl
            << endl;
    }
    if (verbose > 1)
    {
        cout << "Complex Image" << endl;
        cout << complex_image;
        cout << endl
            << endl;
    }

    //Perform 2D fft on image
    cout << "BEFORE here";
    fft2D(complex_image, false, balance, threads, verbose);
    cout << "After fft2d here";
    if (verbose == 1)
    {
        cout << "Performing FFT on Image" << endl;
        ///cout << complex_image;
        cout << endl
            << endl;
    }

    //Threshold the fft

    // for (int i = 0; i < image_M.rows; ++i)
    //     for (int j = 0; j < image_M.cols; ++j)
    //         image_M.at<uint>(i, j) = image[i][j];

    double maximum_value = 0.0;
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            maximum_value = max(maximum_value, abs(complex_image[i][j]));
        }
    }
    threshold *= maximum_value;
    // cout << "threshold :" << threshold << endl;
    int count = 0;

    // Setting values less than threshold to zero
    // This step is responsible for compression
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            if (abs(complex_image[i][j]) < threshold)
            {
                count++;
                complex_image[i][j] = 0;
            }
        }
    }
    int zeros_count = 0;
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            if (abs(complex_image[i][j]) == 0)
            {
                zeros_count++;
            }
        }
    }
    cout << "Components removed(percent): " << ((zeros_count*1.00/(complex_image.size()*complex_image[0].size())))*100 << endl;
    if (verbose > 1)
    {
        cout << "Thresholded Image" << endl;
        //cout << complex_image;
        cout << endl
            << endl;
    }

    // Perform inverse FFT
    fft2D(complex_image, true, balance, threads, verbose);
    if (verbose > 1)
    {
        cout << "Inverted Image" << endl;
        //cout << complex_image;
        cout << endl
            << endl;
    }
    //Convert to uint8 format
    // We will consider only the real part of the image
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            image[i][j] = uint(complex_image[i][j].real() + 0.5);
        }
    }
    if (verbose > 0)
    {
        cout << "Compressed Image" << endl;
        //cout << image;
    }
}


void fft3(vector<vector<vector<base>>> &a, bool invert, bool balance, int threads) {
    int nx = a.size();
    int ny = a[0].size();
    int nz = a[0][0].size();

    // Perform 2D FFT on each slice along the z dimension
    for (int z = 0; z < nz; z++) {
        vector<vector<base>> slice(nx, vector<base>(ny));
        for (int x = 0; x < nx; x++) {
            for (int y = 0; y < ny; y++) {
                slice[x][y] = a[x][y][z];
            }
        }
        fft2D(slice, invert, balance, threads);
        for (int x = 0; x < nx; x++) {
            for (int y = 0; y < ny; y++) {
                a[x][y][z] = slice[x][y];
            }
        }
    }

    // Transpose the data so that the z dimension becomes the y dimension
    vector<vector<vector<base>>> transposed(nz, vector<vector<base>>(nx, vector<base>(ny)));
    for (int x = 0; x < nx; x++) {
        for (int y = 0; y < ny; y++) {
            for (int z = 0; z < nz; z++) {
                transposed[z][x][y] = a[x][y][z];
            }
        }
    }

    // Perform 2D FFT on each slice along the new z dimension
    for (int z = 0; z < nz; z++) {
        fft2D(transposed[z], invert, balance, threads);
    }

    // Transpose the data back to its original orientation
    for (int x = 0; x < nx; x++) {
        for (int y = 0; y < ny; y++) {
            for (int z = 0; z < nz; z++) {
                a[x][y][z] = transposed[z][x][y];
            }
        }
    }
}



void write2DVectorToFile(const std::vector<std::vector<uint>>& data, const std::string& filename) {
    std::ofstream file(filename);

    if (file.is_open()) {
        for (const auto& row : data) {
            for (const auto& element : row) {
                file << element << ' ';
            }
            file << '\n';
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }
}



#define N 100000
#define BALANCE 1024


vector<vector<uint>> read_2d_vector(const std::string& file_path) {
    vector<vector<uint>> data;
    std::ifstream file(file_path);
    std::string line;

    while (std::getline(file, line)) {
        vector<uint> row;
        std::stringstream ss(line);
        uint value;

        while (ss >> value) {
            row.push_back(value);
        }

        data.push_back(row);
    }

    return data;
}

bool areEqual(const vector<vector<vector<base>>>& vec1, const vector<vector<vector<base>>>& vec2) {
    if (vec1.size() != vec2.size()) {
        return false;
    }

    for (size_t i = 0; i < vec1.size(); ++i) {
        if (vec1[i].size() != vec2[i].size()) {
            return false;
        }

        for (size_t j = 0; j < vec1[i].size(); ++j) {
            if (vec1[i][j].size() != vec2[i][j].size()) {
                return false;
            }

            for (size_t k = 0; k < vec1[i][j].size(); ++k) {
                if (abs(vec1[i][j][k] - vec2[i][j][k]) > 1e-6) {
                    return false;
                }
            }
        }
    }

    return true;
}


int main()
{
    // cv::Mat image_M;
    // image_M = imread("squirrel.jpg", IMREAD_GRAYSCALE);
    // if (!image_M.data)
    // {
    //     cout << "Could not open or find the image" << std::endl;
    //     return -1;
    // }
    

    // cv::imwrite("original.jpg", image_M);
    // vector<vector<uint>> image(image_M.rows, vector<uint>(image_M.cols));
    // for (int i = 0; i < image_M.rows; ++i)
    //     for (int j = 0; j < image_M.cols; ++j)
    //         image[i][j] = uint(image_M.at<uint>(i, j));

    // auto temp_image = image;

    // std::vector<std::vector<uint>> image = read_2d_vector("image2d.txt");

    // write2DVectorToFile(image, "originalImage.txt");
    
    // // fft.compress_image(image, 0.00005, 0);
    // freopen("out.txt", "w", stdout);
    // for(double thresh = 0.000001; thresh < 1; thresh*=10)
    // {
    //     cout << "For thresh= " << thresh << endl;
    //     compress_image(image, thresh, BALANCE, 10, 0);
    //     // for (int i = 0; i < image_M.rows; ++i)
    //     //     for (int j = 0; j < image_M.cols; ++j)
    //     //         image_M.at<uint>(i, j) = image[i][j];
    //     // string s = "compressed_";
    //     // s = s+to_string(thresh);
    //     // s += ".jpg";
    //     // cv::imwrite(s, image_M);
        
    //     //call write2DVectorToFile function to write to a file having filename based on threshold
    //     write2DVectorToFile(image, "compressed_image_"+to_string(thresh)+".txt");
    // }
    // Initialize 3D vector with sample data

    int nx = 10, ny = 10, nz = 10;  // replace with your actual dimensions
    vector<vector<vector<base>>> data(nx, vector<vector<base>>(ny, vector<base>(nz)));
    vector<vector<vector<base>>> dataOrig(nx, vector<vector<base>>(ny, vector<base>(nz)));

    // Fill the 3D vector with your sample data
    for (int x = 0; x < nx; x++) {
        for (int y = 0; y < ny; y++) {
            for (int z = 0; z < nz; z++) {
                // replace with your actual data
                data[x][y][z] = 1.0;
                dataOrig[x][y][z] = 1.0;
            }
        }
    }

    // Invoke the fft3 function
    bool invert = false; 
    bool balance = false;  
    int threads = 4;  
    fft3(data, invert, balance, threads);

    // perform inverse FFT3
    fft3(data, true, balance, threads);

    // Check the result
    bool correct = true;
    double epsilon = 1e-1; // or a value that is acceptable in your context
    for (int x = 0; x < nx; x++) {
        for (int y = 0; y < ny; y++) {
            for (int z = 0; z < nz; z++) {
                cout << data[x][y][z] << " ";
                if (abs(data[x][y][z] - dataOrig[x][y][z]) > epsilon) {
                    correct = false;
                    // break;
                }
            }
            // if (!correct) break;
        }
        // if (!correct) break;
    }

    if (correct) {
        cout << "FFT3 is working correctly." << endl;
    } else {
        cout << "FFT3 is not working correctly." << endl;
    }


    return 0;
}

