#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <opencv2/core.hpp>
// #include <opencv2/highgui/highgui.hpp>
#include <chrono> 

#include <opencv4/opencv2/core/core_c.h>
#include <opencv2/highgui/highgui_c.h>
#include <hip/hip_runtime.h>
// #include <crt/hip/device_functions.h>
#include <>
#include <hip/hip_complex.h>
#include <opencv4/opencv2/imgcodecs.hpp>
// #include <opencv2/core/mat.hpp>
// #include <opencv2>s


using namespace std::chrono;
using namespace std;
using namespace cv;

typedef complex<float> base;
typedef float2 Complex_my;

template <typename T>
ostream &operator<<(ostream &o, vector<T> v)
{
    if (v.size() > 0)
        o << v[0];
    for (unsigned i = 1; i < v.size(); i++)
        o << " " << v[i];
    return o << endl;
}
static __device__ __host__ inline Complex_my Add(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x + B.x;
    C.y = A.y + B.y;
    return C;
}

/**
 *  Inverse of Complex_my Number
 */
static __device__ __host__ inline Complex_my Inverse(Complex_my A)
{
    Complex_my C;
    C.x = -A.x;
    C.y = -A.y;
    return C;
}

/**
 *  Multipication of Complex_my Numbers
 */
static __device__ __host__ inline Complex_my Multiply(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x * B.x - A.y * B.y;
    C.y = A.y * B.x + A.x * B.y;
    return C;
}

/**
* Parallel Functions for performing various tasks
*/

/**
*  Dividing by constant for inverse fft transform
*/
__global__ void inplace_divide_invert(Complex_my *A, int n, int threads)
{
    int i = blockIdx.x * threads + threadIdx.x;
    if (i < n)
    {
        // printf("in divide");
        A[i].x /= n;
        A[i].y /= n;
    }
    else
    {
        // printf("else in divide");
        // printf("i=%d, n=%d", i, n);
    }
}

/**
* Reorders array by bit-reversing the indexes.
*/
__global__ void bitrev_reorder(Complex_my *__restrict__ r, Complex_my *__restrict__ d, int s, size_t nthr, int n)
{
    int id = blockIdx.x * nthr + threadIdx.x;
    //r[id].x = -1;
    if (id < n and __brev(id) >> (32 - s) < n)
        r[__brev(id) >> (32 - s)] = d[id];
}

/**
* Inner part of the for loop
*/
__device__ void inplace_fft_inner(Complex_my *__restrict__ A, int i, int j, int len, int n, bool invert)
{
    if (i + j + len / 2 < n and j < len / 2)
    {
        Complex_my u, v;

        float angle = (2 * M_PI * j) / (len * (invert ? 1.0 : -1.0));
        v.x = cos(angle);
        v.y = sin(angle);

        u = A[i + j];
        v = Multiply(A[i + j + len / 2], v);
        // printf("i:%d j:%d u_x:%f u_y:%f    v_x:%f v_y:%f\n", i, j, u.x, u.y, v.x, v.y);
        A[i + j] = Add(u, v);
        A[i + j + len / 2] = Add(u, Inverse(v));
    }
}

// CUDA Kernel function to generate the Hanning window
__global__ void hanningWindow(float *d_in, int N, float scale_factor = 1.0) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float scale = 2.0f * 3.14159265358979323846f / (N - 1);
        d_in[idx] *= scale_factor * 0.5f * (1.0f - cosf(scale * idx));
    }
}

/**
* FFT if number of threads are sufficient.
*/
__global__ void inplace_fft(Complex_my *__restrict__ A, int i, int len, int n, int threads, bool invert)
{
    int j = blockIdx.x * threads + threadIdx.x;
    inplace_fft_inner(A, i, j, len, n, invert);
}

/**
* FFt if number of threads are not sufficient.
*/
__global__ void inplace_fft_outer(Complex_my *__restrict__ A, int len, int n, int threads, bool invert)
{
    int i = (blockIdx.x * threads + threadIdx.x)*len;
    for (int j = 0; j < len / 2; j++)
    {
        inplace_fft_inner(A, i, j, len, n, invert);
    }
}

/**
* parallel FFT transform and inverse transform
* Arguments vector of complex numbers, invert, balance, number of threads
* Perform inplace transform
*/
void fft(vector<base> &a, bool invert, int balance = 10, int threads = 32)
{
    // Creating array from vector
    int n = (int)a.size();
    int data_size = n * sizeof(Complex_my);
    Complex_my *data_array = (Complex_my *)malloc(data_size);
    for (int i = 0; i < n; i++)
    {
        data_array[i].x = a[i].real();
        data_array[i].y = a[i].imag();
    }
    
    // Copying data to GPU
    Complex_my *A, *dn;
    hipMalloc((void **)&A, data_size);
    hipMalloc((void **)&dn, data_size);
    hipMemcpy(dn, data_array, data_size, hipMemcpyHostToDevice);
    // Bit reversal reordering
    int s = log2(n);

    bitrev_reorder<<<ceil(float(n) / threads), threads>>>(A, dn, s, threads, n);

    
    // Synchronize
    hipDeviceSynchronize();
    // Iterative FFT with loop parallelism balancing
    for (int len = 2; len <= n; len <<= 1)
    {
        if (n / len > balance)
        {

            inplace_fft_outer<<<ceil((float)n / threads / len), threads>>>(A, len, n, threads, invert);
        }
        else
        {
            for (int i = 0; i < n; i += len)
            {
                float repeats = len / 2;
                inplace_fft<<<ceil(repeats / threads), threads>>>(A, i, len, n, threads, invert);
            }
        }
    }
    
    if (invert)
        inplace_divide_invert<<<ceil(n * 1.00 / threads), threads>>>(A, n, threads);

    // Copy data from GPU
    Complex_my *result;
    result = (Complex_my *)malloc(data_size);
    hipMemcpy(result, A, data_size, hipMemcpyDeviceToHost);
    
    // Saving data to vector<complex> in input.
    for (int i = 0; i < n; i++)
    {
        a[i] = base(result[i].x, result[i].y);
    }
    // Free the memory blocks
    free(data_array);
    hipFree(A);
    hipFree(dn);
    return;
}

/** Applying hanning window*/
void applyHanningWindow(vector<base> &a) {
    float *d_in;
    hipMalloc(&d_in, a.size() * sizeof(base));
    hipMemcpy(d_in, a.data(), a.size() * sizeof(base), hipMemcpyHostToDevice);

    hanningWindow<<<(a.size() + 255) / 256, 256>>>(d_in, a.size(), 0.7);
    hipDeviceSynchronize();

    hipMemcpy(a.data(), d_in, a.size() * sizeof(base), hipMemcpyDeviceToHost);
    hipFree(d_in);
}

/**
* Performs 2D FFT 
* takes vector of complex vectors, invert and verbose as argument
* performs inplace FFT transform on input vector
*/
void fft2D(vector<vector<base>> &a, bool invert, int balance, int threads, int verbose = 0)
{
    auto matrix = a;
    // Transform the rows
    if (verbose > 0)
        cout << "Transforming Rows" << endl;

    for (auto i = 0; i < matrix.size(); i++)
    {
        //cout<<i<<endl;
        applyHanningWindow(matrix[i]);
        fft(matrix[i], invert, balance, threads);
    }

    // preparing for transforming columns

    if (verbose > 0)
        cout << "Converting Rows to Columns" << endl;

    a = matrix;
    matrix.resize(a[0].size());
    for (int i = 0; i < matrix.size(); i++)
        matrix[i].resize(a.size());

    // Transposing matrix
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            matrix[j][i] = a[i][j];
        }
    }
    if (verbose > 0)
        cout << "Transforming Columns" << endl;

    // Transform the columns
    for (auto i = 0; i < matrix.size(); i++){
        applyHanningWindow(matrix[i]);
        fft(matrix[i], invert, balance, threads);
    }

    if (verbose > 0)
        cout << "Storing the result" << endl;

    // Storing the result after transposing
    // [j][i] is getting value of [i][j]
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            a[j][i] = matrix[i][j];
        }
    }
}

/**
* Function to multiply two polynomial
* takes two polynomials represented as vectors as input
* return the product of two vectors
*/
vector<int> mult(vector<int> a, vector<int> b, int balance, int threads)
{
    // Creating complex vector from input vectors
    vector<base> fa(a.begin(), a.end()), fb(b.begin(), b.end());

    // Padding with zero to make their size equal to power of 2
    size_t n = 1;
    while (n < max(a.size(), b.size()))
        n <<= 1;
    n <<= 1;

    fa.resize(n), fb.resize(n);

    // Transforming both a and b
    // Converting to points form
    fft(fa, false, balance, threads), fft(fb, false, balance, threads);

    // performing point wise multipication of points
    for (size_t i = 0; i < n; ++i)
        fa[i] *= fb[i];

    // Performing Inverse transform
    fft(fa, true, balance, threads);

    // Saving the real part as it will be the result
    vector<int> res;
    res.resize(n);
    for (size_t i = 0; i < n; ++i)
        res[i] = int(fa[i].real() + 0.5);

    return res;
}

/**
* Function to perform jpeg compression on image
* takes image, threshold, verbose as input
* image is represented as vector<vector>
* perform inplace compression on the input
*/
void compress_image(vector<vector<uint>> &image, double threshold, int balance, int threads, int verbose = 1)
{
    //Convert image to complex type
    cout << "here";
    vector<vector<base>> complex_image(image.size(), vector<base>(image[0].size()));
    for (auto i = 0; i < image.size(); i++)
    {
        for (auto j = 0; j < image[0].size(); j++)
        {
            complex_image[i][j] = image[i][j];
        }
    }
    if (verbose == 1)
    {
        cout << "input Image" << endl;
        //cout << image;
        cout << endl
            << endl;
    }
    if (verbose > 1)
    {
        cout << "Complex Image" << endl;
        cout << complex_image;
        cout << endl
            << endl;
    }

    //Perform 2D fft on image
    cout << "BEFORE here";
    fft2D(complex_image, false, balance, threads, verbose);
    cout << "After fft2d here";
    if (verbose == 1)
    {
        cout << "Performing FFT on Image" << endl;
        ///cout << complex_image;
        cout << endl
            << endl;
    }

    //Threshold the fft

    // for (int i = 0; i < image_M.rows; ++i)
    //     for (int j = 0; j < image_M.cols; ++j)
    //         image_M.at<uint>(i, j) = image[i][j];

    double maximum_value = 0.0;
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            maximum_value = max(maximum_value, abs(complex_image[i][j]));
        }
    }
    threshold *= maximum_value;
    // cout << "threshold :" << threshold << endl;
    int count = 0;

    // Setting values less than threshold to zero
    // This step is responsible for compression
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            if (abs(complex_image[i][j]) < threshold)
            {
                count++;
                complex_image[i][j] = 0;
            }
        }
    }
    int zeros_count = 0;
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            if (abs(complex_image[i][j]) == 0)
            {
                zeros_count++;
            }
        }
    }
    cout << "Components removed(percent): " << ((zeros_count*1.00/(complex_image.size()*complex_image[0].size())))*100 << endl;
    if (verbose > 1)
    {
        cout << "Thresholded Image" << endl;
        //cout << complex_image;
        cout << endl
            << endl;
    }

    // Perform inverse FFT
    fft2D(complex_image, true, balance, threads, verbose);
    if (verbose > 1)
    {
        cout << "Inverted Image" << endl;
        //cout << complex_image;
        cout << endl
            << endl;
    }
    //Convert to uint8 format
    // We will consider only the real part of the image
    for (int i = 0; i < complex_image.size(); i++)
    {
        for (int j = 0; j < complex_image[0].size(); j++)
        {
            image[i][j] = uint(complex_image[i][j].real() + 0.5);
        }
    }
    if (verbose > 0)
    {
        cout << "Compressed Image" << endl;
        //cout << image;
    }
}



void write2DVectorToFile(const std::vector<std::vector<uint>>& data, const std::string& filename) {
    std::ofstream file(filename);

    if (file.is_open()) {
        for (const auto& row : data) {
            for (const auto& element : row) {
                file << element << ' ';
            }
            file << '\n';
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }
}



#define N 100000
#define BALANCE 1024


vector<vector<uint>> read_2d_vector(const std::string& file_path) {
    vector<vector<uint>> data;
    std::ifstream file(file_path);
    std::string line;

    while (std::getline(file, line)) {
        vector<uint> row;
        std::stringstream ss(line);
        uint value;

        while (ss >> value) {
            row.push_back(value);
        }

        data.push_back(row);
    }

    return data;
}


int main()
{
    const int size = 100;
    const double pi = 3.14159265358979323846;
    const double frequency1 = 0.1;
    const double frequency2 = 0.2;

    std::vector<std::vector<base>> data(size, std::vector<base>(size));

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            data[i][j] = sin(2 * pi * frequency1 * i) + sin(2 * pi * frequency2 * j);
        }
    }

    // Perform the FFT
    fft2D(data, false, BALANCE, 32, 0);

    std::vector<std::vector<uint>> new_data(size, std::vector<uint>(size));
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            // static cast base to unit before assigning to new_data
            new_data[i][j] = static_cast<uint>(data[i][j].real());
            // new_data[i][j] = data[i][j];
        }
    }

    // Write the data to a file
    write2DVectorToFile(new_data, "dataFFT.txt");

    // Now `data` contains a 2D signal composed of two sine waves of different frequencies
    // You can perform your harmonic analysis on `data`

    return 0;
}

